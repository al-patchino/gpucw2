#include "hip/hip_runtime.h"
#include "matrix_kernel_1.h"
#include <hip/hip_runtime.h>

static __global__ void normaliseRow(int pivotPos, float *d_Matrix);
static __global__ void scaleAndSubtract(int row2_ID, int pivotPos, int width, float *d_Matrix);
static __global__ void scaleAndSubtract2(int row2_ID, int pivotPos, int width, float *d_Matrix);

/*
This is the naive implementation of the host algorithm.

	* Since it only uses one block, then it may only do matrices of width < 1024.

*/

// -- Controller function for device function
void M1_Controller(float* d_Matrix, float* h_Matrix, int height, int width){

	// Iterate through all rows
	for (int row_ID = 0; row_ID < height; row_ID++){

		// Each row normalisation has fewer non-zero elements to normalise
		dim3 blocksPerGrid(1);
		dim3 threadsPerBlock(width - row_ID);

		// -- Define pivot position and value
		int pivotPos = row_ID * (width + 1);

		//printf("pivotPos %d\n", pivotPos);

		// -- Normalise row
		//printf("Lauching normaliseRow<<<%d, %d>>>...\n", blocksPerGrid.x, threadsPerBlock.x);
		normaliseRow <<<blocksPerGrid, threadsPerBlock >>>(pivotPos, d_Matrix);

		// -- Wait for kernel to finish normalising row
		hipDeviceSynchronize();
		
		// -- Loop through j-th column and remove suitable multiples
		for (int row2_ID = 1; row2_ID < (height - row_ID); row2_ID++){

			// -- Each row will have few non-zero elements to remove
			dim3 blocksPerGrid(1);
			dim3 threadsPerBlock(width - row_ID);

			//-- Call kernel to scale and subtract rows
			scaleAndSubtract <<< blocksPerGrid, threadsPerBlock >> >(row2_ID, pivotPos, width, d_Matrix);
	
		}	
	}

	// -- Go through all rows starting from second
	for (int row_ID = 1; row_ID < height; row_ID++){

		// -- Define pivot position and value
		int pivotPos = row_ID * (width + 1);

		// -- Loop through j-th column and remove suitable multiples
		for (int row2_ID = 1; row2_ID < (row_ID + 1); row2_ID++){

			// -- Each row will have few non-zero elements to remove
			dim3 blocksPerGrid(1);
			dim3 threadsPerBlock(width - row_ID);

			//-- Call kernel to scale and subtract rows
			scaleAndSubtract2 << < blocksPerGrid, threadsPerBlock >> >(row2_ID, pivotPos, width, d_Matrix);

		}
	}

}

// -- Normalise row relative to pivot value
__global__ void normaliseRow(int pivotPos, float *d_Matrix){

	// -- Get threadID
	int tid = threadIdx.x;

	// -- Normalise element relative to pivot value
	d_Matrix[pivotPos + tid] = d_Matrix[pivotPos + tid] / d_Matrix[pivotPos];
}

// -- Normalise row relative to pivot value downwards
__global__ void scaleAndSubtract(int row2_ID, int pivotPos, int width, float *d_Matrix){

	// -- Get threadID
	int tid = threadIdx.x;

	// -- Find coefficient to scale row with
	float coeff = d_Matrix[pivotPos + (width * row2_ID)];

	// -- Update elements in row by subtracting elements with coeff
	d_Matrix[pivotPos + (width * row2_ID) + tid] = d_Matrix[pivotPos + (width * row2_ID) + tid]
		- (coeff * d_Matrix[pivotPos + tid]);
}

// -- Normalise row relative to pivot value upwards
__global__ void scaleAndSubtract2(int row2_ID, int pivotPos, int width, float *d_Matrix){

	// -- Get threadID
	int tid = threadIdx.x;

	// -- Find coefficient to scale row with
	float coeff = d_Matrix[pivotPos - (width * row2_ID)];

	// -- Update elements in row by subtracting elements with coeff
	d_Matrix[pivotPos - (width * row2_ID) + tid] = d_Matrix[pivotPos - (width * row2_ID) + tid]
		- (coeff * d_Matrix[pivotPos + tid]);
}


